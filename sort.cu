#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N = 1000000
//This merge just 
__global__ void compare_and_swap(double * smaller, double larger){
    
}

int main(){
    double *arr;
    double *d_arr;
    arr = (double*) malloc(sizeof(double)*N);
    hipMalloc((void**)&d_arr, sizeof(double)*N);
    //Initialize the array values...
    //TODO:
    //Transfer the memory
    //TODO:
    //call the merge. threads will have their own id. for each iteration,
    //we also pass some value n which is the size of the array.
    for(int i = 1;i <N;i = i*2){
        //First, calculate the TOTAL number of threads that we think we need. This will be N/i...
        int totalThreads = N/i;
        int threadsPerBlock = min(1024, totalThreads);
        if (threadsPerBlock == 1024){
            int blocks = (totalThreads+1023)/1024;
            merge<<<blocks,1024>>>(d_arr, i);
        }
        else
            merge<<<1, threadsPerBlock>>>(d_arr, i);
    }
}


int min(int a, int b) {
    return (a < b) ? a : b;
}