
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<=n)
        out[tid] = a[tid] + b[tid];
}


int main(){
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Allocate GPU memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Transfer data to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    vector_add<<<N/256,256>>>(d_out, d_a, d_b, N);

    ///sync the device before grabbing memory
    hipDeviceSynchronize();

    //return the out array
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    //cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    printf("%f\n",out[1]);
    
    //more cleanup
    free(a);
    free(b);
    free(out);
}
